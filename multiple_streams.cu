
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <vector>
#include <array>

#define CHECK_CUDA(func)  {                                                \
  hipError_t status = (func);                                             \
  if (status != hipSuccess) {                                             \
    printf("CUDA API failed at line %d with error: %s\n",                  \
           __LINE__, hipGetErrorString(status));                          \
    return EXIT_FAILURE;                                                   \
  }                                                                         \
}

#define CHECK_CUSPARSE(func)  {                                             \
  hipsparseStatus_t status = (func);                                         \
  if (status != HIPSPARSE_STATUS_SUCCESS) {                                  \
    printf("CUSPARSE API failed at line %d with error.\n",                  \
           __LINE__);                                                       \
    return EXIT_FAILURE;                                                    \
  }                                                                         \
}


struct MatrixData {
    int num_matrices;
    std::vector<int> num_rows, num_cols, nnz;
    std::vector<std::array<int, 9>> h_rows, h_columns;
    std::vector<std::array<float, 9>> h_values;
    std::vector<std::array<float, 4>> hX, hY, hY_result;
};

MatrixData initialize_matrices() {
    MatrixData data;
    data.num_matrices = 3;
    data.num_rows = {4, 4, 4};
    data.num_cols = {4, 4, 4};
    data.nnz = {9, 9, 9};
    data.h_rows = {{
        {0, 0, 0, 1, 2, 2, 2, 3, 3},
        {0, 0, 0, 1, 2, 2, 2, 3, 3},
        {0, 0, 0, 1, 2, 2, 2, 3, 3}
    }};
    data.h_columns = {{
        {0, 2, 3, 1, 0, 2, 3, 1, 3},
        {0, 2, 3, 1, 0, 2, 3, 1, 3},
        {0, 2, 3, 1, 0, 2, 3, 1, 3}
    }};
    data.h_values = {{
        {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f},
        {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f},
        {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f}
    }};
    data.hX = {{
        {1.0f, 2.0f, 3.0f, 4.0f},
        {1.0f, 2.0f, 3.0f, 4.0f},
        {1.0f, 2.0f, 3.0f, 4.0f}
    }};
    data.hY = {{
        {0.0f, 0.0f, 0.0f, 0.0f},
        {0.0f, 0.0f, 0.0f, 0.0f},
        {0.0f, 0.0f, 0.0f, 0.0f}
    }};
    data.hY_result = {{
        {19.0f, 8.0f, 51.0f, 52.0f},
        {19.0f, 8.0f, 51.0f, 52.0f},
        {19.0f, 8.0f, 51.0f, 52.0f}
    }};

    return data;
}


int main(void) {
  // Define your problem and inputs (matrix and vectors) here.

  // Host problem definition
  float alpha = 1.0f;
  float beta = 0.0f;

  MatrixData data = initialize_matrices();

  auto num_matrices = data.num_matrices;
  auto num_rows = data.num_rows;
  auto num_cols = data.num_cols;
  auto nnz = data.nnz;
  auto h_rows = data.h_rows;
  auto h_columns = data.h_columns;
  auto h_values = data.h_values;
  auto hX = data.hX;
  auto hY = data.hY;
  auto hY_result = data.hY_result;


// Define device arrays/vectors to hold matrices and vector data
std::vector<int*> dA_rows(num_matrices), dA_columns(num_matrices);
std::vector<float*> dA_values(num_matrices), dX(num_matrices), dY(num_matrices);

// Create arrays of streams and cusparseHandles
std::vector<hipStream_t> streams(num_matrices);
std::vector<hipsparseHandle_t> handles(num_matrices);
std::vector<hipsparseSpMatDescr_t> matA(num_matrices);
std::vector<hipsparseDnVecDescr_t> vecX(num_matrices);
std::vector<hipsparseDnVecDescr_t> vecY(num_matrices);

std::vector<void*> dBuffers(num_matrices);
std::vector<size_t> bufferSize(num_matrices);

// Allocate device memory for each matrix and vector
for (auto i = 0; i < num_matrices; i++) {
    CHECK_CUDA(hipMalloc((void**) &dA_rows[i], nnz[i] * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**) &dA_columns[i], nnz[i] * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**) &dA_values[i], nnz[i] * sizeof(float)))
    CHECK_CUDA(hipMalloc((void**) &dX[i], num_cols[i] * sizeof(float)))
    CHECK_CUDA(hipMalloc((void**) &dY[i], num_rows[i] * sizeof(float)))
    CHECK_CUDA(hipMalloc(&dBuffers[i], bufferSize[i]) )
}

for (int i = 0; i < num_matrices; i++) {

    hipStreamCreate(&streams[i]);
    hipsparseCreate(&handles[i]);
    hipsparseSetStream(handles[i], streams[i]);

// Copy matrices and vectors from host to device asynchronously
    CHECK_CUDA(hipMemcpyAsync(dA_rows[i], h_rows[i].data(), nnz[i] * sizeof(int), hipMemcpyHostToDevice, streams[i]))
    CHECK_CUDA(hipMemcpyAsync(dA_columns[i], h_columns[i].data(), nnz[i] * sizeof(int), hipMemcpyHostToDevice, streams[i]))
    CHECK_CUDA(hipMemcpyAsync(dA_values[i], h_values[i].data(), nnz[i] * sizeof(float), hipMemcpyHostToDevice, streams[i]))
    CHECK_CUDA(hipMemcpyAsync(dX[i], hX[i].data(), num_cols[i] * sizeof(float), hipMemcpyHostToDevice, streams[i]))
    CHECK_CUDA(hipMemcpyAsync(dY[i], hY[i].data(), num_rows[i] * sizeof(float), hipMemcpyHostToDevice, streams[i]))

    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCoo(&matA[i], num_rows[i], num_cols[i], nnz[i],
                                      dA_rows[i], dA_columns[i], dA_values[i],
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX[i], num_cols[i], dX[i], HIP_R_32F) )

    // Create dense vector Y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY[i], num_rows[i], dY[i], HIP_R_32F) )

    // Allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                    handles[i], HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA[i], vecX[i], &beta, vecY[i], HIP_R_32F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize[i]) )


    CHECK_CUSPARSE( hipsparseSpMV(
                                    handles[i], HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha, matA[i], vecX[i], &beta, vecY[i], HIP_R_32F,
                                    HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA[i]) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX[i]) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY[i]) )

    CHECK_CUDA(hipMemcpyAsync(hY[i].data(), dY[i], num_rows[i] * sizeof(float), hipMemcpyDeviceToHost, streams[i]))
}


  // Synchronize each stream
  for (auto &stream : streams) {
    hipStreamSynchronize(stream);
  }

  for (int i = 0; i < num_matrices; i++) {

    int correct = 1;
    for (int j = 0; j < num_rows[i]; j++) {
        if (hY[i][j] != hY_result[i][j]) { // direct floating point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("spmv_coo_example test PASSED\n");
    else
        printf("spmv_coo_example test FAILED: wrong result\n");
  }

  // Destroy handles and streams
  for(auto &handle : handles) {
    hipsparseDestroy(handle);
  }

  for(auto &stream : streams) {
    hipStreamDestroy(stream);
  }

  return 0;
}
